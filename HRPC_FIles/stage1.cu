#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>


// ---------------------------------------------------------------------------- 
// CUDA code to compute minimun distance between n points
//
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAX_POINTS 1048576

// ---------------------------------------------------------------------------- 
// Kernel Function to compute distance between all pairs of points
// Input: 
//	X: X[i] = x-coordinate of the ith point
//	Y: Y[i] = y-coordinate of the ith point
//	n: number of points
// Output: 
//	D: D[0] = minimum distance
//
__global__ void minimum_distance(float* X, float* Y, volatile float* D, int n) {

    // ------------------------------------------------------------
    //
    // Kernel function code goes here
    //
    // ------------------------------------------------------------
}
// ---------------------------------------------------------------------------- 
// Host function to compute minimum distance between points
// Input:
//	X: X[i] = x-coordinate of the ith point
//	Y: Y[i] = y-coordinate of the ith point
//	n: number of points
// Output: 
//	D: minimum distance
//
float minimum_distance_host(float* X, float* Y, int n) {
    float dx, dy, Dij, min_distance, min_distance_i;
    int i, j;
    dx = X[1] - X[0];
    dy = Y[1] - Y[0];
    min_distance = sqrtf(dx * dx + dy * dy);
    for (i = 0; i < n - 1; i++) {
        for (j = i + 1; j < i + 2; j++) {
            dx = X[j] - X[i];
            dy = Y[j] - Y[i];
            min_distance_i = sqrtf(dx * dx + dy * dy);
        }
        for (j = i + 1; j < n; j++) {
            dx = X[j] - X[i];
            dy = Y[j] - Y[i];
            Dij = sqrtf(dx * dx + dy * dy);
            if (min_distance_i > Dij) min_distance_i = Dij;
        }
        if (min_distance > min_distance_i) min_distance = min_distance_i;
    }
    return min_distance;
}
// ---------------------------------------------------------------------------- 
// Print device properties
void print_device_properties() {
    int i, deviceCount;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&deviceCount);
    printf("------------------------------------------------------------\n");
    printf("Number of GPU devices found = %d\n", deviceCount);
    for (i = 0; i < deviceCount; ++i) {
        hipGetDeviceProperties(&deviceProp, i);
        printf("[Device: %1d] Compute Capability %d.%d.\n", i, deviceProp.major, deviceProp.minor);
        printf(" ... multiprocessor count  = %d\n", deviceProp.multiProcessorCount);
        printf(" ... max threads per multiprocessor = %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf(" ... max threads per block = %d\n", deviceProp.maxThreadsPerBlock);
        printf(" ... max block dimension   = %d, %d, %d (along x, y, z)\n",
            deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf(" ... max grid size         = %d, %d, %d (along x, y, z)\n",
            deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf(" ... warp size             = %d\n", deviceProp.warpSize);
        printf(" ... clock rate            = %d MHz\n", deviceProp.clockRate / 1000);
    }
    printf("------------------------------------------------------------\n");
}
// ---------------------------------------------------------------------------- 
// Main program - initializes points and computes minimum distance 
// between the points
//
int main(int argc, char* argv[]) {

    // Host Data
    float* hVx;		// host x-coordinate array
    float* hVy;		// host y-coordinate array
    float hmin_dist;		// minimum value on host

    // Device Data
    float* dVx;		// device x-coordinate array
    float* dVy;		// device x-coordinate array
    float* dmin_dist;		// minimum value on device

    // Device parameters
    int MAX_BLOCK_SIZE;		// Maximum number of threads allowed on the device
    int blocks;			// Number of blocks in grid
    int threads_per_block;	// Number of threads per block

    // Timing variables
    hipEvent_t start, stop;		// GPU timing variables
    struct timespec cpu_start, cpu_stop; // CPU timing variables
    float time_array[10];

    // Other variables
    int i, size, num_points;
    float min_distance, sqrtn;
    int seed = 0;

    // Print device properties
    print_device_properties();

    // Get device information and set device to use
    int deviceCount;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount > 0) {
        hipSetDevice(0);
        hipGetDeviceProperties(&deviceProp, 0);
        MAX_BLOCK_SIZE = deviceProp.maxThreadsPerBlock;
    }
    else {
        printf("Warning: No GPU device found ... results may be incorrect\n");
    }

    // Timing initializations
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Check input
    if (argc != 2) {
        printf("Use: %s <number of points>\n", argv[0]);
        exit(0);
    }
    if ((num_points = atoi(argv[argc - 1])) < 2) {
        printf("Minimum number of points allowed: 2\n");
        exit(0);
    }
    if ((num_points = atoi(argv[argc - 1])) > MAX_POINTS) {
        printf("Maximum number of points allowed: %d\n", MAX_POINTS);
        exit(0);
    }

    // Allocate host coordinate arrays 
    size = num_points * sizeof(float);
    hVx = (float*)malloc(size);
    hVy = (float*)malloc(size);

    // Initialize points
    srand(seed);
    sqrtn = (float)sqrt(num_points);
    for (i = 0; i < num_points; i++) {
        hVx[i] = sqrtn * (float)rand();
        hVy[i] = sqrtn * (float)rand();
    }

    // Allocate device coordinate arrays
    hipMalloc(&dVx, size);
    hipMalloc(&dVy, size);
    hipMalloc(&dmin_dist, size);

    // Copy coordinate arrays from host memory to device memory 
    hipEventRecord(start, 0);

    hipMemcpy(dVx, hVx, size, hipMemcpyHostToDevice);
    hipMemcpy(dVy, hVy, size, hipMemcpyHostToDevice);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[0]), start, stop);

    // Invoke kernel
    hipEventRecord(start, 0);

    // ------------------------------------------------------------
    //
    // Invoke kernel function(s) here
    //
    // ------------------------------------------------------------

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[1]), start, stop);

    // Copy result from device memory to host memory 
    hipEventRecord(start, 0);

    hipMemcpy(&hmin_dist, dmin_dist, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[2]), start, stop);

    // Compute minimum distance on host to check device computation
    // clock_gettime(CLOCK_REALTIME, &cpu_start);

    min_distance = minimum_distance_host(hVx, hVy, num_points);

    // clock_gettime(CLOCK_REALTIME, &cpu_stop);
    time_array[3] = 1000 * ((cpu_stop.tv_sec - cpu_start.tv_sec)
        + 0.000000001 * (cpu_stop.tv_nsec - cpu_start.tv_nsec));

    // Print results
    printf("Number of Points    = %d\n", num_points);
    printf("GPU Host-to-device  = %f ms \n", time_array[0]);
    printf("GPU Device-to-host  = %f ms \n", time_array[2]);
    printf("GPU execution time  = %f ms \n", time_array[1]);
    printf("CPU execution time  = %f ms\n", time_array[3]);
    printf("Min. distance (GPU) = %e\n", hmin_dist);
    printf("Min. distance (CPU) = %e\n", min_distance);
    printf("Relative error      = %e\n", fabs(min_distance - hmin_dist) / min_distance);


    // Free device memory 
    hipFree(dVx);
    hipFree(dVy);
    hipFree(dmin_dist);

    // Free host memory 
    free(hVx);
    free(hVy);
}
